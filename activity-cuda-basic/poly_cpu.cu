#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>


// Global  Host Code Function called polly_gpu_exponational

_global_ void poly_gpu_ex (float* poly, int degree, int n, float* array) {

    // Generates Index based on blockid ,Dim and Thread IDS
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  // Find Polynomial Value and add it to array at Index.
  if( index < n )
  {
    float finalPoli = 0.0;
    float powerOf = 1.0;
    for ( int i = 0; i < degree+1; ++i)
    {
      finalPoli += powerOf * poly[i];
      powerOf *= array[index];
    }
    array[index] = finalPoli;
}
}


int main (int argc, char* argv[]) {
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

// Sequential
  int n = atoi(argv[1]);
  int degree = atoi(argv[2]);
  int nbiter = atoi(argv[3]);

  float* array = new float[n];
  for (int i=0; i<n; ++i)
    array[i] = 1.;
  float* poly = new float[degree+1];
  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  float *dev_array, *dev_poly;
  
  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
	
	hipMallocManaged(&dev_array, n*sizeof(float));
  	hipMallocManaged(&dev_poly, (degree+1)*sizeof(float));
  
  	hipMemcpy(dev_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(dev_poly, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice);

  	poly_gpu_ex<<<(n+255)/256, 256>>>(dev_poly, degree, n, dev_array);
  	hipMemcpy(array, dev_array, n*sizeof(float), hipMemcpyDeviceToHost);
       
        hipFree(dev_array);
        hipFree(dev_poly);

        hipDeviceSynchronize();

  end = std::chrono::system_clock::now();array[0]+=1;
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;
  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = true;
	ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect . Please Check .
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}